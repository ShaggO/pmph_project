#include "hip/hip_runtime.h"
#ifndef PROJ_HELPER_FUNS_CU
#define PROJ_HELPER_FUNS_CU

#include "ProjHelperFun.cu.h"
#include "ProjKernels.cu.h"
#include "TridagKernel.cu.h"

/**************************/
/**** HELPER FUNCTIONS ****/
/**************************/

// Copy triple-nested vector togpu device array
void cpCpu2Gpu(
        vector<vector<vector<REAL > > >& src,
        unsigned numX, unsigned numY, unsigned numZ,
        REAL* dst) {
    int mem_size = sizeof(REAL)*numX*numY*numZ;
    // Allocate local flat array
    REAL* local = (REAL*) malloc(mem_size);
    for (int i = 0; i < numX; i++) {
        for (int j = 0; j < numY; j++) {
            // Copy each subarray into the flat array
            memcpy((void*) &local[i*numY*numZ+j*numZ],(void*) &src[i][j][0], sizeof(REAL)*numZ);
        }
    }
    // Copy flat array to device
    hipMemcpy(dst,local,mem_size,hipMemcpyHostToDevice);
    free(local);
}

// Copy glob vector to gpu device glob
void cpGlob2Gpu(
        vector<PrivGlobs>& globs,
        unsigned outer,
        unsigned numX,
        unsigned numY,
        unsigned numT,
        DevicePrivGlobs &d_globs) {
    // Allocate local flat array
    REAL* myX =         (REAL*) malloc(sizeof(REAL)*outer*numX);
    REAL* myY =         (REAL*) malloc(sizeof(REAL)*outer*numY);
    REAL* myTimeline =  (REAL*) malloc(sizeof(REAL)*outer*numT);
    REAL* myResult =    (REAL*) malloc(sizeof(REAL)*outer*numX*numY);
    REAL* myVarX =      (REAL*) malloc(sizeof(REAL)*outer*numX*numY);
    REAL* myVarY =      (REAL*) malloc(sizeof(REAL)*outer*numX*numY);
    REAL* myDxx =       (REAL*) malloc(sizeof(REAL)*outer*numX*4);
    REAL* myDyy =       (REAL*) malloc(sizeof(REAL)*outer*numY*4);

    // Copy each subarray into the flat arrays
    for (int i = 0; i < outer; i++) {
        memcpy(&myX[i*numX],&globs[i].myX[0],sizeof(REAL)*numX);
        memcpy(&myY[i*numY],&globs[i].myY[0],sizeof(REAL)*numY);
        memcpy(&myTimeline[i*numT],&globs[i].myTimeline[0],sizeof(REAL)*numT);
        for (int j = 0; j < numX; j++) {
            memcpy(&myResult[i*numX*numY+j*numY],&globs[i].myResult[j][0], sizeof(REAL)*numY);
            memcpy(&myVarX[i*numX*numY+j*numY],&globs[i].myVarX[j][0], sizeof(REAL)*numY);
            memcpy(&myVarY[i*numX*numY+j*numY],&globs[i].myVarY[j][0], sizeof(REAL)*numY);
            memcpy(&myDxx[i*numX*4+j*4],&globs[i].myDxx[j][0],sizeof(REAL)*4);
        }
        for (int j = 0; j < numY; j++) {
            memcpy(&myDyy[i*numY*4+j*4],&globs[i].myDyy[j][0],sizeof(REAL)*4);
        }
    }

    // Copy flat array to device
    hipMemcpy(d_globs.myX,myX,sizeof(REAL)*outer*numX,hipMemcpyHostToDevice);
    hipMemcpy(d_globs.myY,myY,sizeof(REAL)*outer*numY,hipMemcpyHostToDevice);
    hipMemcpy(d_globs.myTimeline,myTimeline,sizeof(REAL)*outer*numT,hipMemcpyHostToDevice);
    hipMemcpy(d_globs.myResult,myResult,sizeof(REAL)*outer*numX*numY,hipMemcpyHostToDevice);
    hipMemcpy(d_globs.myVarX,myVarX,sizeof(REAL)*outer*numX*numY,hipMemcpyHostToDevice);
    hipMemcpy(d_globs.myVarY,myVarY,sizeof(REAL)*outer*numX*numY,hipMemcpyHostToDevice);
    hipMemcpy(d_globs.myDxx,myDxx,sizeof(REAL)*outer*numX*4,hipMemcpyHostToDevice);
    hipMemcpy(d_globs.myDyy,myDyy,sizeof(REAL)*outer*numY*4,hipMemcpyHostToDevice);
    d_globs.myXindex = globs[0].myXindex;
    d_globs.myYindex = globs[0].myYindex;

    // Clean up
    free(myX);      free(myY);      free(myTimeline);
    free(myResult); free(myVarX);   free(myVarY);
    free(myDxx);    free(myDyy);
}
/**
 * Fills in:
 *   globs.myTimeline  of size [0..numT-1]
 *   globs.myX         of size [0..numX-1]
 *   globs.myY         of size [0..numY-1]
 * and also sets
 *   globs.myXindex and globs.myYindex (both scalars)
 */
void initGrid(  const REAL s0, const REAL alpha, const REAL nu,const REAL t,
                const unsigned numX, const unsigned numY, const unsigned numT, PrivGlobs& globs
) {
    for(unsigned i=0;i<numT;++i)
        globs.myTimeline[i] = t*i/(numT-1);

    const REAL stdX = 20.0*alpha*s0*sqrt(t);
    const REAL dx = stdX/numX;
    globs.myXindex = static_cast<unsigned>(s0/dx) % numX;

    for(unsigned i=0;i<numX;++i)
        globs.myX[i] = i*dx - globs.myXindex*dx + s0;

    const REAL stdY = 10.0*nu*sqrt(t);
    const REAL dy = stdY/numY;
    const REAL logAlpha = log(alpha);
    globs.myYindex = static_cast<unsigned>(numY/2.0);

    for(unsigned i=0;i<numY;++i)
        globs.myY[i] = i*dy - globs.myYindex*dy + logAlpha;
}

/**
 * Fills in:
 *    Dx  [0..n-1][0..3] and
 *    Dxx [0..n-1][0..3]
 * Based on the values of x,
 * Where x's size is n.
 */
void initOperator(  const vector<REAL>& x,
                    vector<vector<REAL> >& Dxx
) {
	const unsigned n = x.size();

	REAL dxl, dxu;

	//	lower boundary
	//dxl		 =  0.0;
	//dxu		 =  x[1] - x[0];

	Dxx[0][0] =  0.0;
	Dxx[0][1] =  0.0;
	Dxx[0][2] =  0.0;
    Dxx[0][3] =  0.0;

	//	standard case
	for(unsigned i=1;i<n-1;i++)
	{
		dxl      = x[i]   - x[i-1];
		dxu      = x[i+1] - x[i];

		Dxx[i][0] =  2.0/dxl/(dxl+dxu);
		Dxx[i][1] = -2.0*(1.0/dxl + 1.0/dxu)/(dxl+dxu);
		Dxx[i][2] =  2.0/dxu/(dxl+dxu);
        Dxx[i][3] =  0.0;
	}

	//	upper boundary
	//dxl		   =  x[n-1] - x[n-2];
	//dxu		   =  0.0;

	Dxx[n-1][0] = 0.0;
	Dxx[n-1][1] = 0.0;
	Dxx[n-1][2] = 0.0;
    Dxx[n-1][3] = 0.0;
}

/*********************/
/*** Tridag Kernel ***/
/*********************/
// Try to optimize it: for example,
//    (The allocated shared memory is enough for 8 floats / thread):
//    1. the shared memory space for "mat_sh" can be reused for "lin_sh"
//    2. with 1., now you have space to hold "u" and "uu" in shared memory.
//    3. you may hold "a[gid]" in a register, since it is accessed twice, etc.
__global__ void
TRIDAG_SOLVER(  REAL* a,
                REAL* b,
                REAL* c,
                REAL* r,
                const unsigned int n,
                const unsigned int sgm_sz,
                REAL* u,
                REAL* uu
) {
    const unsigned int tid = threadIdx.x;
    const unsigned int gid = blockIdx.x*blockDim.x + tid;

    // total shared memory (declared outside)
    extern __shared__ char sh_mem[];
    // shared memory space for the 2x2 matrix multiplication SCAN
    volatile MyReal4* mat_sh = (volatile MyReal4*)sh_mem;
    // shared memory space for the linear-function composition SCAN
    // reuses mat_sh memory as it is calculated after mat_sh
    volatile MyReal2* lin_sh = (volatile MyReal2*) (mat_sh);
    // shared memory space for the flag array
    volatile int*     flg_sh = (volatile int*    ) (mat_sh + blockDim.x);
    volatile REAL*    uu_sh  = (volatile REAL*   ) (flg_sh + blockDim.x);
    volatile REAL*     u_sh  = (volatile REAL*   ) (uu_sh  + blockDim.x);

    // make the flag array
    flg_sh[tid] = (tid % sgm_sz == 0) ? 1 : 0;
    __syncthreads();

    REAL agid = -a[gid];
    //--------------------------------------------------
    // Recurrence 1: b[i] = b[i] - a[i]*c[i-1]/b[i-1] --
    //   solved by scan with 2x2 matrix mult operator --
    //--------------------------------------------------
    // 1.a) first map
    const unsigned int beg_seg_ind = (gid / sgm_sz) * sgm_sz;
    const unsigned int begseg = (tid / sgm_sz) * sgm_sz;
    REAL b0 = (gid < n) ? b[beg_seg_ind] : 1.0;
    mat_sh[tid] = (gid!=beg_seg_ind && gid < n) ?
                    MyReal4(b[gid], agid*c[gid-1], 1.0, 0.0) :
                    MyReal4(1.0,                 0.0, 0.0, 1.0) ;
    // 1.b) inplaceScanInc<MatMult2b2>(n,mats);
    __syncthreads();
    MyReal4 res4 = sgmScanIncBlock <MatMult2b2, MyReal4, int>(mat_sh, flg_sh, tid);
    // 1.c) second map
    if(gid < n) {
        uu_sh[tid] = (res4.x*b0 + res4.y) / (res4.z*b0 + res4.w) ;
        //uu[gid] = (res4.x*b0 + res4.y) / (res4.z*b0 + res4.w) ;
    }
    __syncthreads();

    // make the flag array
    flg_sh[tid] = (tid % sgm_sz == 0) ? 1 : 0;
    __syncthreads();

    //----------------------------------------------------
    // Recurrence 2: y[i] = y[i] - (a[i]/b[i-1])*y[i-1] --
    //   solved by scan with linear func comp operator  --
    //----------------------------------------------------
    // 2.a) first map
    REAL y0 = (gid < n) ? r[beg_seg_ind] : 1.0;
    lin_sh[tid] = (gid!=beg_seg_ind && gid < n) ?
                    MyReal2(r[gid], agid/uu_sh[tid-1]) :
                    MyReal2(0.0,    1.0              ) ;
    // 2.b) inplaceScanInc<LinFunComp>(n,lfuns);
    __syncthreads();
    MyReal2 res2 = sgmScanIncBlock <LinFunComp, MyReal2, int>(lin_sh, flg_sh, tid);
    // 2.c) second map
    if(gid < n) {
        u_sh[tid] = res2.x + y0*res2.y;
        //u[gid] =  res2.x + y0*res2.y;
    }
    __syncthreads();

    // make the flag array
    flg_sh[tid] = (tid % sgm_sz == 0) ? 1 : 0;
    __syncthreads();
#if 1
    //----------------------------------------------------
    // Recurrence 3: backward recurrence solved via     --
    //             scan with linear func comp operator  --
    //----------------------------------------------------
    // 3.a) first map
    const unsigned int end_seg_ind = (beg_seg_ind + sgm_sz) - 1;
    const unsigned int k = (end_seg_ind - gid) + beg_seg_ind ;
    const unsigned int endseg = begseg + sgm_sz - 1;
    const unsigned int ksh = endseg - tid + begseg;
    REAL yn = u_sh[endseg] / uu_sh[endseg];
    lin_sh[tid] = (gid!=beg_seg_ind && gid < n) ?
                    MyReal2( u_sh[ksh]/uu_sh[ksh], -c[k]/uu_sh[ksh] ) :
                    MyReal2( 0.0,        1.0         ) ;
    // 3.b) inplaceScanInc<LinFunComp>(n,lfuns);
    __syncthreads();
    MyReal2 res3 = sgmScanIncBlock <LinFunComp, MyReal2, int>(lin_sh, flg_sh, tid);
    __syncthreads();
    // 3.c) second map
    if(gid < n) {
        u[k] = res3.x + yn*res3.y;
    }
#endif
}
#endif //PROJ_HELPER_FUN
